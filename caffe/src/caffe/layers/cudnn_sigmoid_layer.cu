
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNSigmoidLayer<Dtype,Mtype>::Forward_gpu(const vector<Blob<Dtype,Mtype>*>& bottom,
    const vector<Blob<Dtype,Mtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnActivationForward(Caffe::cudnn_handle(),
        CUDNN_ACTIVATION_SIGMOID,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
}

template <typename Dtype, typename Mtype>
void CuDNNSigmoidLayer<Dtype,Mtype>::Backward_gpu(const vector<Blob<Dtype,Mtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype,Mtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnActivationBackward(Caffe::cudnn_handle(),
        CUDNN_ACTIVATION_SIGMOID,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSigmoidLayer);


}  // namespace caffe
#endif
